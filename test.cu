#include "hip/hip_runtime.h"
//
// Created by steve on 17-3-9.
//

#include <iostream>
//#include "stdio.h"
//#include "hip/hip_runtime.h"
#include "/usr/include/hip/hip_runtime.h"
#include "../../../../usr/include/host_defines.h"
//#include "../../../../usr/include///hip/hip_runtime_api.h"
//#include "../../../../usr/include/c++/6/cstdio"

__global__ void test_add(int * a , int * b,int *c)
{
    int ii= threadIdx.x;
    c[ii] = a[ii]+b[ii];
    return;
}

int main()
{
    int **t;
    hipMalloc((void**)(&t),100*sizeof(int*));
//    printf("in the function");
    std::cout << "in function " << std::endl;
    int a[10],b[10],c[10];
    for(int i(0);i<10;++i)
    {
        a[i] = i*2;
        b[i]=i*10;
        std::cout << "a,b:"<<a[i]<<"  "<<b[i]<<std::endl;
    }
    int *da,*db,*dc;
    hipMalloc((void**)&da,10*sizeof(int*));
    hipMalloc((void**)&db,10*sizeof(int*));
    hipMalloc((void**)&dc,10*sizeof(int*));
    hipMemcpy(da,a,10*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,b,10*sizeof(int),hipMemcpyHostToDevice);

    test_add<<<1,10>>>(da,db,dc);
    hipMemcpy(c,dc,10*sizeof(int),hipMemcpyDeviceToHost);
    for(int i(0);i<10;++i)
    {
        std::cout << "c:"<<i<<":"<<c[i]<<std::endl;
    };




    return 0;
}